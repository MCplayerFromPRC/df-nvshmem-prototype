#include "hip/hip_runtime.h"
/****
 * Copyright (c) 2014, NVIDIA Corporation.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *
 *    * Redistributions of source code must retain the above copyright notice,
 *      this list of conditions and the following disclaimer.
 *    * Redistributions in binary form must reproduce the above copyright
 *      notice, this list of conditions and the following disclaimer in the
 *      documentation and/or other materials provided with the distribution.
 *    * Neither the name of the NVIDIA Corporation nor the names of its
 *      contributors may be used to endorse or promote products derived from
 *      this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF
 * THE POSSIBILITY OF SUCH DAMAGE.
 ****/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "nvshmem.h"
#include "nvshmem_device.h"
#include "mpi.h"

#define CUDA_CHECK(stmt)                                \
do {                                                    \
    hipError_t result = (stmt);                        \
    if (hipSuccess != result) {                        \
        fprintf(stderr, "[%s:%d] cuda failed with %s \n",   \
         __FILE__, __LINE__,hipGetErrorString(result));\
        exit(-1);                                       \
    }                                                   \
    assert(hipSuccess == result);                      \
} while (0)

__global__ void ping_pong (int *data_d, int *flag_d, int len, int pe, int iter, int skip) {
    long long int start, stop, time; 
    int i, peer;
 
    peer = !pe;   

    for (i=0; i<(iter+skip); i++) {
       if (i == skip) start = clock64(); 
 
       //printf("pe: %d peer: %d cond: %d \n", pe, peer, i&1);

       if ((i&1) == pe) { 
            nvshmem_int_wait_until (flag_d, 0, i+1);

	    nvshmem_int_put (data_d, data_d, len, peer);
            nvshmem_quiet ();   
	    nvshmem_int_p (flag_d, i+1, peer);
            nvshmem_quiet ();   
       } else {
	    nvshmem_int_put (data_d, data_d, len, peer);
            nvshmem_quiet ();   
	    nvshmem_int_p (flag_d, i+1, peer);
            nvshmem_quiet ();   

            nvshmem_int_wait_until (flag_d, 0, i+1);
       }
    }
    stop = clock64();

    if (pe == 0) { 
        time = (stop - start)/iter;
        printf("%7d \t %lld \n", len*sizeof(int), time);
    }
}

int main (int c, char *v[])
{
    int local_rank = 0;
    int dev_count, mype, npes, size; 
    int *flag_d, *data_d; 

    int iter = 200; 
    int skip = 20;
    int max_msg_size = 64*1024;

    CUDA_CHECK(hipGetDeviceCount(&dev_count));
    if (dev_count <= 0) {
        fprintf(stderr, "no CUDA devices found \n");
        exit(-1);
    }

    if (getenv("MV2_COMM_WORLD_LOCAL_RANK") != NULL) { 
        local_rank = atoi(getenv("MV2_COMM_WORLD_LOCAL_RANK"));   
    }
    CUDA_CHECK(hipSetDevice(local_rank%dev_count));

    MPI_Init (&c, &v);
    nvstart_pes();
    mype = nvshmem_my_pe();
    npes = nvshmem_n_pes();

    if (npes != 2) { 
        fprintf(stderr, "This test requires exactly two processes \n");
        goto finalize;
    }
 
    data_d = (int *) nvshmalloc (max_msg_size); 
    flag_d = (int *) nvshmalloc (sizeof(int)); 
    hipMemset(data_d, 0, sizeof(int));
    hipMemset(flag_d, 0, sizeof(int));
    CUDA_CHECK(hipDeviceSynchronize());

    if (mype == 0) { 
       printf("   size \t latency(cycles)\n", time);
       fflush(stdout);
    }

    for (size = sizeof(int); size <= max_msg_size; size *= 2) {  
        ping_pong <<<1, 1>>> (data_d, flag_d, size/sizeof(int), mype, iter, skip);  

        nvshmem_barrier_all();
    }

finalize:

    nvshmcleanup();

    nvstop_pes();
    MPI_Finalize();

    return 0;
}
